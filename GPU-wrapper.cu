#include "includes/GPU_wrapper.h"
#include "includes/kernel.cuh"

/******************************************************************************/
// helper functions

void print_martix(big* b,int rows,int cols){
    for(int i = 0;i<rows;i++){
    for(int j = 0;j<cols;j++)
        std::cout << b[i*cols + j].x << " ";
    std::cout << std::endl;
    }
    cout << endl;
}

big bighToBig(bigH g){
    uint AND = UINT32_MAX;
    big q;
    uint64_t word = g.lower();
    q.x = word&AND;
    word >>= 32;
    q.y = word&AND;
    word = g.upper();
    q.z = word&AND;
    word >>= 32;
    q.w = word&AND;
    return q;
}

bigH bigTobigH(big q){
    bigH g(0);
    g += q.w;
    g <<= 32;
    g += q.z;
    g <<= 32;
    g += q.y;
    g <<= 32;
    g += q.x;
    return g;
}

big* convert(bigH* matrix,long size){
    big* ret = (big *)malloc(sizeof(big)*size);
    for(long i =0;i<size;i++){
        ret[i] = bighToBig(matrix[i]);
    }
    return ret;
}

bigH* convertBack(big* matrix,long size){
    bigH* ret = (bigH *)malloc(sizeof(bigH)*size);
    for(long i =0;i<size;i++){
        ret[i] = bigTobigH(matrix[i]);
    }
    return ret;
}


void encryptHelper(big* A,big* R,big* G,big* result,big q,uint bits,unsigned char bit,int n,int m){

    unsigned int grid_rows = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    gpu_sparse_mult<<<dimGrid,dimBlock>>>(A,R,result,n,m,bits,q);
    hipDeviceSynchronize();
    if(bit == 1){
        int blockSize = 256;
        int numBlocks = (m*n + blockSize - 1) / blockSize;
        gpu_add<<<numBlocks, blockSize>>>(n*m,q, G, result);
    }
}

void fillRandom(big q, uint bits, big* R,long size){ // will work only for modulus size strictly less than 128
    
    long words = size/32;
    long rem = size%32;
    uint t = 1,arr[4];
    t <<= rem;

    arr[0] = 0;arr[1] = 0;arr[2] = 0;arr[3] = 0;

    for(long i = 0;i<size;i++){
        for(long j = 0;j<words;j++){
            arr[j] = rand();
        }
        arr[words] = rand()%t;
        R[i].x = arr[0];R[i].y = arr[1];R[i].z = arr[2];R[i].w = arr[3];
        // sub_cpu(&R[i],q);
    }

}
/******************************************************************************/
// Tests

void test(big* A,big* R,big* result,big q,uint bits,uint n,uint m){
    big* d_matrix1,*d_matrix2,*d_result;
    hipMalloc((void **)&d_matrix1,sizeof(u128)*n*m);
    hipMalloc((void **)&d_matrix2,sizeof(u128)*n*m);
    hipMalloc((void **)&d_result,sizeof(u128)*n*m);

    hipMemcpy(d_matrix1,A,sizeof(u128)*m*n,hipMemcpyHostToDevice);
    hipMemcpy(d_matrix2,R,sizeof(u128)*m*n,hipMemcpyHostToDevice);

    unsigned int grid_rows = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    gpu_sparse_mult<<<dimGrid,dimBlock>>>(d_matrix1,d_matrix2,d_result,n,m,bits,q);

    big* chck = (big* )malloc(sizeof(big)*m*n);
    hipMemcpy(chck,d_result,sizeof(u128)*m*n,hipMemcpyDeviceToHost);

    bool pass = 1;
    for(int i = 0;i<m*n;i++){
        if(chck[i].x != result[i].x || chck[i].y != result[i].y || chck[i].z != result[i].z || chck[i].w != result[i].w){
            cout << "test failed\n\n";
            pass = 0;
            break;
        }
    }

    if(pass) cout << "test passed\n";

    hipFree(d_result);
    hipFree(d_matrix1);
    hipFree(d_matrix2);

    free(chck);
}

void MAIN_TEST_GPU(bigH* A_h,bigH* R_h,bigH* result_h,bigH g,uint bits,int n,int m){
    big* A = convert(A_h,m*n);
    big* R = convert(R_h,m*n);
    big* result = convert(result_h,m*n);

    big q = bighToBig(g);

    test(A,R,result,q,bits,n,m);
    
    free(A);
    free(R);
    free(result);
    
}

void test_generator(bigH g,uint bits){
    big q = bighToBig(g);
}

/******************************************************************************/


bigH* encrypt(bigH* pk_h,bigH* R_h,bigH* G_h,bigH q_h,uint n,uint m,uint bits,unsigned char bit){
    long size = m*n;
    big* PK = convert(pk_h,size);
    big* R = convert(R_h,size);
    big* G = convert(G_h,size);
    big g = bighToBig(q_h);

    
    big* pk_d,*R_d,*G_d,*result_d;
    hipMalloc((void **)&pk_d,sizeof(big)*size);
    hipMalloc((void **)&R_d,sizeof(big)*size);
    hipMalloc((void **)&G_d,sizeof(big)*size);
    hipMalloc((void **)&result_d,sizeof(big)*size);

    hipMemcpy(pk_d,PK,sizeof(u128)*size,hipMemcpyHostToDevice);
    hipMemcpy(R_d,R,sizeof(u128)*size,hipMemcpyHostToDevice);
    hipMemcpy(G_d,G,sizeof(u128)*size,hipMemcpyHostToDevice);

    free(PK);
    free(G);

    encryptHelper(pk_d,R_d,G_d,result_d,g,bits,bit,n,m);

    hipMemcpy(R,result_d,sizeof(u128)*size,hipMemcpyDeviceToHost);;
    bigH* cipherText = convertBack(R,size);


    hipFree(pk_d);
    hipFree(R_d);
    hipFree(G_d);
    hipFree(result_d);

    return cipherText;

}


